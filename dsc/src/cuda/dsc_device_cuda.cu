#include "hip/hip_runtime.h"
// Copyright (c) 2024, Christian Gilli <christian.gilli@dspcraft.com>
// All rights reserved.
//
// This code is licensed under the terms of the 3-clause BSD license
// (https://opensource.org/license/bsd-3-clause).

#include "dsc_device.h"
#include "cuda/dsc_cuda.h"

#define DSC_DEVICE_CUDA_ALIGN ((usize) 1024)
#define DSC_MEMCPY_DIRECTIONS ((int) 3)

static constexpr hipMemcpyKind DSC_CUDA_MEMCPY_DIRECTIONS[DSC_MEMCPY_DIRECTIONS] = {
    hipMemcpyDeviceToHost,
    hipMemcpyHostToDevice,
    hipMemcpyDeviceToDevice,
};

static DSC_CUDA_KERNEL void k_init_random(hiprandState *state) {
    DSC_CUDA_TID();
    hiprand_init(clock64(), tid, 0, &state[tid]);
}

static void cuda_memcpy(void *dst, const void *src, const usize nb, const dsc_memcpy_dir dir) {
    DSC_CUDA_FAIL_ON_ERROR(hipMemcpy(dst, src, nb, DSC_CUDA_MEMCPY_DIRECTIONS[dir]));
}

static void cuda_dispose(dsc_device *dev) {
    DSC_CUDA_FAIL_ON_ERROR(hipFree(dev->device_mem));

    const dsc_cuda_dev_info *info = (dsc_cuda_dev_info *) dev->extra_info;

    DSC_LOG_INFO("%s:%d device %s disposed",
                 DSC_DEVICE_NAMES[dev->type],
                 info->dev_idx,
                 info->name);
}

dsc_device *dsc_cuda_device(const usize mem_size, const int cuda_dev) {
    static dsc_cuda_dev_info extra = {
        .name = {},
        .randState = {},
        .dev_idx = cuda_dev,
    };

    static dsc_device dev = {
        .used_nodes = {},
        .free_nodes = {},
        .head = {},
        .device_mem = {},
        .extra_info = &extra,
        .mem_size = DSC_ALIGN(mem_size, DSC_DEVICE_CUDA_ALIGN),
        .used_mem = 0,
        .type = CUDA,
        .memcpy = cuda_memcpy,
        .dispose = cuda_dispose,
    };

    DSC_CUDA_FAIL_ON_ERROR(hipSetDevice(cuda_dev));

    dsc_cuda_dev_name(cuda_dev, extra.name);

    DSC_CUDA_FAIL_ON_ERROR(hipMalloc(&extra.randState, DSC_CUDA_DEFAULT_THREADS * sizeof(hiprandState)));

    k_init_random<<<1, DSC_CUDA_DEFAULT_THREADS>>>(extra.randState);

    DSC_CUDA_FAIL_ON_ERROR(hipDeviceSynchronize());

    DSC_CUDA_FAIL_ON_ERROR(hipMalloc(&dev.device_mem, dev.mem_size));

    dev.free_nodes[0].size = dev.mem_size;
    dev.free_nodes[0].data = dev.device_mem;
    dev.free_nodes[0].next = nullptr;

    dev.head = &dev.free_nodes[0];

    DSC_LOG_INFO("%s:%d device %s initialized with a buffer of %ldMB (total: %ldMB)",
                 DSC_DEVICE_NAMES[dev.type],
                 cuda_dev,
                 extra.name,
                 (usize) DSC_B_TO_MB(dev.mem_size),
                 (usize) DSC_B_TO_MB(dsc_cuda_dev_mem(cuda_dev)));

    return &dev;
}